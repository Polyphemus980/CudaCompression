#include "hip/hip_runtime.h"
#include "fixed-length_kernels.cuh"

static constexpr int BITS_IN_BYTE = 8;
static constexpr int FRAME_LENGTH = 128;
static constexpr int THREADS_PER_BLOCK = 1024;
static constexpr int FRAMES_PER_BLOCK = THREADS_PER_BLOCK / FRAME_LENGTH;

struct multiply_and_add {
	const uint32_t multiplier; 

	multiply_and_add(int _multiplier) : multiplier(_multiplier) {}

	__host__ __device__
		uint32_t operator()(uint32_t prev_sum, uint32_t current_value) const {
		return prev_sum + multiplier * current_value;
	}
};

__device__ int countLeadingZeros(unsigned char data) {
	if (data == 0)
		return 7;
	int count = 0;
	while ((data & (1 << 7)) == 0)
	{
		count++;
		data <<= 1;
	}
	return count;
}
__global__ void calculateFrameBits(unsigned char* data,int length,uint32_t* frameBits){
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (threadId >= length)
		return; 

	__shared__ int maxes[FRAMES_PER_BLOCK];
	if (threadIdx.x < FRAMES_PER_BLOCK)
		maxes[threadIdx.x] = 0;
	
	__syncthreads();

	int bits = BITS_IN_BYTE - countLeadingZeros(data[threadId]);
	atomicMax(&maxes[threadIdx.x / FRAME_LENGTH], bits);

	__syncthreads();

	int elements_in_block = min(length - blockIdx.x * blockDim.x, blockDim.x);
	int frames_used = (elements_in_block + FRAME_LENGTH - 1) / FRAME_LENGTH;

	if (threadIdx.x < frames_used) {
		frameBits[blockIdx.x * FRAMES_PER_BLOCK + threadIdx.x] = maxes[threadIdx.x];
	}
}


__global__ void fillOutput(unsigned char* data, int length, uint32_t* frameBits, uint32_t* output,int* framePositions) {
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= length)
		return;

	int frameIndex = threadId / FRAME_LENGTH;

	uint32_t bitsPerSymbol = frameBits[frameIndex];

	int symbolIndex = threadIdx.x % FRAME_LENGTH;
	int startPos = framePositions[frameIndex] + symbolIndex * bitsPerSymbol;

	int outputIndex = startPos / 32;
	
	int bitOffset = startPos % 32;

	unsigned char symbol = data[threadId];

	uint32_t maskedSymbol = (uint32_t)(symbol & ((1u << bitsPerSymbol) - 1));

	uint32_t shiftedSymbol = maskedSymbol << bitOffset;

	atomicOr(&output[outputIndex], shiftedSymbol);

	if (bitOffset + bitsPerSymbol > 32) {
		uint32_t spillBits = bitOffset + bitsPerSymbol - 32;
		uint32_t spillMask = maskedSymbol >> (bitsPerSymbol - spillBits);
		atomicOr(&output[outputIndex + 1], spillMask);
	}

}

int calculateOutputSize(const int* framePositions, int numFrames, std::vector<uint32_t> frameBits,uint64_t dataLength) {
	int lastFrameStart = framePositions[numFrames - 1];
	int lastFrameBitLength = frameBits[numFrames - 1];
	int lastFrameDataCount = dataLength % FRAME_LENGTH == 0 ? FRAME_LENGTH : dataLength % FRAME_LENGTH;
	int lastBitPosition = lastFrameStart + lastFrameBitLength * lastFrameDataCount;
	return (lastBitPosition + 31) / 32;
}

std::vector<unsigned char> convertToUnsignedChar(std::vector<uint32_t>& input) {
	std::vector<unsigned char> output;
	output.reserve(input.size());

	for (uint32_t value : input) {
		assert(value < 256 && "Value exceeds unsigned char range!");
		output.push_back(static_cast<unsigned char>(value));
	}

	return output;
}

__host__ FLData CudaFLEncode(std::vector<unsigned char> data) {
	unsigned char* dev_data = NULL;
	uint64_t length = data.size();

	hipMalloc((void**)&dev_data, sizeof(char) * length);
	hipMemcpy(dev_data, data.data(), sizeof(char) * length, hipMemcpyHostToDevice);

	int numBlocks = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	int numFrames = (length + FRAME_LENGTH - 1) / FRAME_LENGTH;

	uint32_t* dev_frameBits = NULL;
	hipMalloc((void**)&dev_frameBits, sizeof(uint32_t) * numFrames);

	calculateFrameBits << <numBlocks, THREADS_PER_BLOCK >> > (dev_data, length, dev_frameBits);

	std::vector<uint32_t> host_frameBits(numFrames);
	hipMemcpy(host_frameBits.data(), dev_frameBits, sizeof(uint32_t) * numFrames, hipMemcpyDeviceToHost);

	int* dev_framePositions = NULL;
	hipMalloc((void**)&dev_framePositions, sizeof(int) * numFrames);


	thrust::exclusive_scan(thrust::device, dev_frameBits, dev_frameBits + numFrames, dev_framePositions, 0, multiply_and_add(FRAME_LENGTH));

	int* framePositions = (int*)malloc(sizeof(int) * numFrames);
	hipMemcpy(framePositions, dev_framePositions, sizeof(int) * numFrames, hipMemcpyDeviceToHost);

	uint32_t* dev_output = NULL;
	int outputLength = calculateOutputSize(framePositions, numFrames, host_frameBits, length);
	hipMalloc((void**)&dev_output, sizeof(uint32_t) * outputLength);
	hipMemset(dev_output, 0, sizeof(uint32_t) * outputLength);

	fillOutput << <numBlocks, THREADS_PER_BLOCK >> > (dev_data, length, dev_frameBits, dev_output, dev_framePositions);

	std::vector<uint32_t> output(outputLength);
	hipMemcpy(output.data(), dev_output, sizeof(uint32_t) * outputLength, hipMemcpyDeviceToHost);

	FLData encodedData;

	encodedData.encodedValues = output;
	encodedData.frameBits = convertToUnsignedChar(host_frameBits);
	encodedData.valuesLength = encodedData.encodedValues.size();
	encodedData.bitsLength = encodedData.frameBits.size();
	encodedData.decodedDataLength = data.size();

	return encodedData;
}

__global__ void Decode(uint32_t* encodedData, int encodedLength, unsigned char* frameBits, uint64_t decodedDataLength,unsigned char* decoded, uint32_t* framePositions)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= decodedDataLength)
		return;
	
	int frameIndex = threadId / FRAME_LENGTH;

	int bitsPerSymbol = frameBits[frameIndex];

	int symbolIndex = threadIdx.x % FRAME_LENGTH;
	int startPos = framePositions[frameIndex] + symbolIndex * bitsPerSymbol;
	int outputIndex = startPos / 32;

	int bitOffset = startPos % 32;

	unsigned char data = (encodedData[outputIndex] >> bitOffset);

	if (bitOffset + bitsPerSymbol > 32)
	{
		int spilledBits = bitOffset + bitsPerSymbol - 32;
		unsigned char spilledData = (encodedData[outputIndex + 1] & ((1u << spilledBits) - 1)) << (32 - bitOffset);
		data |= spilledData;
	}

	data &= ((1u << bitsPerSymbol) - 1);
	decoded[threadId] = data;

}

__host__ std::vector<unsigned char> CudaFLDecode(FLData decodingData) {

	uint64_t encodedLength = decodingData.valuesLength;
	uint64_t frameBitsLength = decodingData.bitsLength;
	uint64_t decodedDataLength = decodingData.decodedDataLength;

	uint32_t* dev_encodedData = NULL;
	hipMalloc((void**)&dev_encodedData, sizeof(uint32_t) * encodedLength);
	hipMemcpy(dev_encodedData,decodingData.encodedValues.data(), sizeof(uint32_t) * encodedLength, hipMemcpyHostToDevice);
	
	unsigned char* dev_frameBits = NULL;
	hipMalloc((void**)&dev_frameBits, sizeof(unsigned char) * frameBitsLength);
	hipMemcpy(dev_frameBits,decodingData.frameBits.data(), sizeof(unsigned char) * frameBitsLength, hipMemcpyHostToDevice);

	uint32_t* dev_framePositions = NULL;
	hipMalloc((void**)&dev_framePositions, sizeof(int) * frameBitsLength);

	uint32_t* framePositions = (uint32_t*)malloc(sizeof(uint32_t) * frameBitsLength);
	int sum = 0;
	for (int i = 0; i < frameBitsLength; i++) {
		framePositions[i] = sum;
		sum += decodingData.frameBits[i] * FRAME_LENGTH;
	}

	hipMemcpy(dev_framePositions, framePositions, sizeof(uint32_t) * frameBitsLength, hipMemcpyHostToDevice);
	int numBlocks = (decodedDataLength + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	unsigned char* dev_decoded;
	hipMalloc((void**)&dev_decoded, sizeof(char) * decodedDataLength);
	hipMemset(dev_decoded, 0, sizeof(char) * decodedDataLength);
	hipDeviceSynchronize();
	Decode<<<numBlocks, THREADS_PER_BLOCK >>>(dev_encodedData, encodedLength, dev_frameBits, decodedDataLength, dev_decoded, dev_framePositions);
	hipDeviceSynchronize();
	
	std::vector<unsigned char> host_decoded(decodedDataLength);
	hipMemcpy(host_decoded.data(), dev_decoded, sizeof(char) * decodedDataLength, hipMemcpyDeviceToHost);
	
	return host_decoded;
}